#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "hip/hip_vector_types.h"
#include "cuda_shape.h"
#include "cuda_ray.h"
#include "cuda_camera.h"
#include "cuda_scene.h"
#include "tonemapping.h"
#include "sampling.h"
#include "render_parameters.h"

auto constexpr WIDTH = 640;
auto constexpr HEIGHT = 480;

#define IMG_BLACK make_uchar4(0, 0, 0, 0)

unsigned int wangHash(unsigned int a)
{
    //http://raytracey.blogspot.com/2015/12/gpu-path-tracing-tutorial-2-interactive.html
    a = (a ^ 61) ^ (a >> 16);
    a = a + (a << 3);
    a = a ^ (a >> 4);
    a = a * 0x27d4eb2d;
    a = a ^ (a >> 15);

    return a;
}

__inline__ __device__ void running_estimate(float3& acc_buffer, const float3& curr_est, unsigned int N)
{
    acc_buffer += (curr_est - acc_buffer) / (N + 1.f);
}

struct HitInfo
{
    bool intersected = false;
    float t = FLT_MAX;
    float3 normal;
    unsigned int matID;
};

__global__ void testSimpleScene(uchar4* img, cudaScene scene, RenderParameters params, unsigned int hashed_N)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int idy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int offset = idy * scene.camera.imageW + idx;
    img[offset] = IMG_BLACK;

    hiprandState rng;
    hiprand_init(hashed_N + offset, 0, 0, &rng);
    cudaRay pRay;
    scene.camera.GenerateRay(idx, idy, rng, &pRay);

    float3 L = make_float3(0.f, 0.f, 0.f);
    float3 T = make_float3(1.f, 1.f, 1.f);

    for(auto k = 0; k < 5; ++k)
    {
        HitInfo hit;
        //find nearest intersection
#ifdef __GEO__AABB__
        for(auto i = 0; i < scene.num_aabb_boxes; ++i)
        {
            const cudaAABB& box = scene.aabb_boxes[i];
            float ttmp;
            if(box.Intersect(pRay, &ttmp) && (ttmp < hit.t))
            {
                hit.intersected = true;
                hit.t = ttmp;
                hit.normal = box.GetNormal(pRay.PointOnRay(hit.t));
                hit.matID = box.material_id;
            }
        }
#endif

#ifdef __GEO_SPHERE__
        for(auto i = 0; i < scene.num_spheres; ++i)
        {
            const cudaSphere& sphere = scene.spheres[i];
            float ttmp;
            if(sphere.Intersect(pRay, &ttmp) && (ttmp < hit.t))
            {
                hit.intersected = true;
                hit.t = ttmp;
                hit.normal = sphere.GetNormal(pRay.PointOnRay(hit.t));
                hit.matID = sphere.material_id;
            }
        }
#endif

#ifdef __GEO_PLANE__
        for(auto i = 0; i < scene.num_planes; ++i)
        {
            const cudaPlane& plane = scene.planes[i];
            float ttmp;
            if(plane.Intersect(pRay, &ttmp) && (ttmp < hit.t))
            {
                hit.intersected = true;
                hit.t = ttmp;
                hit.normal = plane.GetNormal(pRay.PointOnRay(hit.t));
                hit.matID = plane.material_id;
            }
        }
#endif

        if(!hit.intersected)
        {
            L = make_float3(0.f);
            break;
        }

        pRay.orig = pRay.PointOnRay(hit.t);
        pRay.dir = cosine_weightd_sample_hemisphere(rng, hit.normal);

        L += scene.materials[hit.matID].emition * T;
        T *= scene.materials[hit.matID].albedo;
    }

    running_estimate(params.hdr_buffer[offset], L, params.iteration_count);
    L = reinhard_tone_mapping(params.hdr_buffer[offset], 0.6f);
    img[offset] = make_uchar4(fabsf(L.x) * 255, fabsf(L.y) * 255, fabsf(L.z) * 255, 0);
}

extern "C" void test(uchar4* img, cudaScene& scene, RenderParameters& params)
{
    dim3 blockSize(16, 16);
    dim3 gridSize(640 / blockSize.x, 480 / blockSize.y);

    if(params.iteration_count == 0)
    {
        checkCudaErrors(hipMemset(params.hdr_buffer, 0, sizeof(float3) * WIDTH * HEIGHT));
    }

    testSimpleScene<<<gridSize, blockSize>>>(img, scene, params, wangHash(params.iteration_count));
    checkCudaErrors(hipDeviceSynchronize());
}