#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "hip/hip_vector_types.h"
#include "cuda_shape.h"
#include "cuda_camera.h"
#include "cuda_scene.h"
#include "tonemapping.h"
#include "render_parameters.h"
#include "kernel_globals.h"
#include "shader.h"

auto constexpr WIDTH = 640;
auto constexpr HEIGHT = 480;

__global__ void testSimpleScene(uchar4* img, cudaScene scene, RenderParameters params, unsigned int hashed_N)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int idy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int offset = idy * scene.camera.imageW + idx;
    img[offset] = IMG_BLACK;

    hiprandState rng;
    hiprand_init(hashed_N + offset, 0, 0, &rng);
    cudaRay ray;
    scene.camera.GenerateRay(idx, idy, rng, &ray);

    float3 L = make_float3(0.f, 0.f, 0.f);
    float3 T = make_float3(1.f, 1.f, 1.f);

    SurfaceElement se;
    //todo: add russian roulette
    for(auto k = 0; k < 10; ++k)
    {
        if(!scene_intersect(scene, ray, se)) break;
        L += T * scene.materials[se.matID].emition;

        switch(scene.materials[se.matID].bsdf_type)
        {
            case BSDF_DIFFUSE:
                diffuse_shading(scene, se, rng, &ray, &T);
                break;
            case BSDF_GLASS:
                refractive_shading(scene, se, rng, &ray, &T);
                break;
            case BSDF_GLOSSY:
                glossy_shading(scene, se, rng, &ray, &T);
                break;
            case BSDF_PLASTIC:
                coat_shading(scene, se, rng, &ray, &T);
                break;
            default:
                break;
        }

        //russian roulette
        if(k >= 3)
        {
            float p = fmaxf(T.x, fmaxf(T.y, T.z));
            if(hiprand_uniform(&rng) > p) break;
            T /= p;
        }
    }

    running_estimate(params.hdr_buffer[offset], L, params.iteration_count);
    L = reinhard_tone_mapping(params.hdr_buffer[offset], params.exposure);
    img[offset] = make_uchar4(fabsf(L.x) * 255, fabsf(L.y) * 255, fabsf(L.z) * 255, 0);
}

extern "C" void test(uchar4* img, cudaScene& scene, RenderParameters& params)
{
    dim3 blockSize(16, 16);
    dim3 gridSize(640 / blockSize.x, 480 / blockSize.y);

    if(params.iteration_count == 0)
    {
        checkCudaErrors(hipMemset(params.hdr_buffer, 0, sizeof(float3) * WIDTH * HEIGHT));
    }

    testSimpleScene<<<gridSize, blockSize>>>(img, scene, params, wangHash(params.iteration_count));
    checkCudaErrors(hipDeviceSynchronize());
}