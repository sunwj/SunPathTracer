#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "hip/hip_vector_types.h"
#include "cuda_shape.h"
#include "cuda_camera.h"
#include "cuda_scene.h"
#include "tonemapping.h"
#include "sampling.h"
#include "render_parameters.h"
#include "kernel_globals.h"

auto constexpr WIDTH = 640;
auto constexpr HEIGHT = 480;

__global__ void testSimpleScene(uchar4* img, cudaScene scene, RenderParameters params, unsigned int hashed_N)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int idy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int offset = idy * scene.camera.imageW + idx;
    img[offset] = IMG_BLACK;

    hiprandState rng;
    hiprand_init(hashed_N + offset, 0, 0, &rng);
    cudaRay ray;
    scene.camera.GenerateRay(idx, idy, rng, &ray);

    float3 L = make_float3(0.f, 0.f, 0.f);
    float3 T = make_float3(1.f, 1.f, 1.f);

    HitInfo hi;
    for(auto k = 0; k < 5; ++k)
    {
        if(!scene_intersect(scene, ray, hi)) break;
        L += T * scene.materials[hi.matID].emition;

        if(scene.materials[hi.matID].bsdf_type == BSDF_DIFFUSE)
        {
            ray.orig = hi.pt;
            ray.dir = cosine_weightd_sample_hemisphere(rng, hi.normal);

            T *= scene.materials[hi.matID].albedo;
        }

        if(scene.materials[hi.matID].bsdf_type == BSDF_GLASS)
        {
            float eta = scene.materials[hi.matID].ior;
            if(dot(hi.normal, ray.dir) > 0.f)
            {
                eta = 1.f / eta;
                hi.normal = -hi.normal;
            }
            eta = 1.f / eta;
            float cosin = -dot(hi.normal, ray.dir);
            float cost2 = 1.f - eta * eta * (1.f - cosin * cosin);

            if(cost2 < 0.f)
            {
                T *= scene.materials[hi.matID].albedo;
                ray.dir = reflect(ray.dir, hi.normal);
            }
            else
            {
                float3 tdir = eta * ray.dir + hi.normal * (eta * cosin - sqrtf(cost2));
                tdir = normalize(tdir);

                float n1 = (cosin < 0.f) ? 1.f : scene.materials[hi.matID].ior;
                float n2 = (cosin < 0.f) ? scene.materials[hi.matID].ior : 1.f;
                float R0 = (n1 - n2) * (n1 - n2) / ((n1 + n2) * (n1 + n2));
                float c = cosin;
                float Pr = R0 + (1.f - R0) * c * c * c * c * c;
                float Pt = 1.f - Pr;
                float P = 0.25f + 0.5f * Pr;

                if(hiprand_uniform(&rng) < P)
                {
                    T *= scene.materials[hi.matID].albedo;
                    T *= (Pr / P);
                    ray.dir = reflect(ray.dir, hi.normal);
                }
                else
                {
                    T *= scene.materials[hi.matID].albedo;
                    T *= (Pt / (1.f - P));
                    ray.dir = tdir;
                }
            }

            ray.orig = hi.pt;
        }
    }

    running_estimate(params.hdr_buffer[offset], L, params.iteration_count);
    L = reinhard_tone_mapping(params.hdr_buffer[offset], 0.6f);
    img[offset] = make_uchar4(fabsf(L.x) * 255, fabsf(L.y) * 255, fabsf(L.z) * 255, 0);
}

extern "C" void test(uchar4* img, cudaScene& scene, RenderParameters& params)
{
    dim3 blockSize(16, 16);
    dim3 gridSize(640 / blockSize.x, 480 / blockSize.y);

    if(params.iteration_count == 0)
    {
        checkCudaErrors(hipMemset(params.hdr_buffer, 0, sizeof(float3) * WIDTH * HEIGHT));
    }

    testSimpleScene<<<gridSize, blockSize>>>(img, scene, params, wangHash(params.iteration_count));
    checkCudaErrors(hipDeviceSynchronize());
}