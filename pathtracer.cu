#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "hip/hip_vector_types.h"
#include "cuda_shape.h"
#include "cuda_ray.h"
#include "cuda_camera.h"
#include "cuda_scene.h"

#define IMG_BLACK make_uchar4(0, 0, 0, 0)

__host__ __device__ unsigned int wangHash(unsigned int a)
{
    //http://raytracey.blogspot.com/2015/12/gpu-path-tracing-tutorial-2-interactive.html
    a = (a ^ 61) ^ (a >> 16);
    a = a + (a << 3);
    a = a ^ (a >> 4);
    a = a * 0x27d4eb2d;
    a = a ^ (a >> 15);

    return a;
}

__global__ void testSimpleScene(uchar4* img, cudaScene scene)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int idy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int offset = idy * scene.camera.imageW + idx;
    img[offset] = IMG_BLACK;

    hiprandState rng;
    hiprand_init(0, 0, 0, &rng);
    cudaRay pRay;
    scene.camera.GenerateRay(idx, idy, rng, &pRay);

    float t = FLT_MAX;
    float3 n = make_float3(0.f);
    for(auto i = 0; i < scene.num_aabb_boxes; ++i)
    {
        const cudaAABB& box = scene.aabb_boxes[i];
        float ttmp;
        if(box.Intersect(pRay, &ttmp) && (ttmp < t))
        {
            t = ttmp;
            n = box.GetNormal(pRay.PointOnRay(t));
            float3 c = scene.materials[box.material_id].reflectance * 1.f;
        }
    }

    for(auto i = 0; i < scene.num_spheres; ++i)
    {
        const cudaSphere& sphere = scene.spheres[i];
        float ttmp;
        if(sphere.Intersect(pRay, &ttmp) && (ttmp < t))
        {
            t = ttmp;
            n = sphere.GetNormal(pRay.PointOnRay(t));
        }
    }

    float costerm = fmaxf(0.f, dot(n, normalize(make_float3(-1.f, 0.5f, 3.f))));
    img[offset] = make_uchar4(fabsf(n.x) * 255 * costerm, fabsf(n.y) * 255 * costerm, fabsf(n.z) * 255 * costerm, 0);
}

extern "C" void test(uchar4* img, cudaScene& scene)
{
    dim3 blockSize(16, 16);
    dim3 gridSize(640 / blockSize.x, 480 / blockSize.y);

    testSimpleScene<<<gridSize, blockSize>>>(img, scene);
    checkCudaErrors(hipDeviceSynchronize());
}